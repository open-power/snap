#include "hip/hip_runtime.h"
#include <kernel.h>
#include <stdint.h>

uint8_t *d_A, *d_B, *d_C;

__global__ void add_uint8(uint8_t *A, uint8_t *B, uint8_t *C, int N){

	int id = blockIdx.x*blockDim.x+threadIdx.x;
	if (id<N){
		C[id] = A[id] + B[id];
	}
}


void cuda_add(void* A, void* B, void* C, int N){

	size_t size = N*sizeof(uint8_t);

	hipMalloc(&d_A, size);
	hipMalloc(&d_B, size);
	hipMalloc(&d_C, size);

	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

	int blockSize = 64;
	int numBlocks = N/64;

	add_uint8<<<numBlocks,blockSize>>>(d_A,d_B,d_C,N);

	hipMemcpy(C,d_C, size, hipMemcpyDeviceToHost);
	
}

void cuda_Finish(){
	
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}
